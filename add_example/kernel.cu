#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <time.h>

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void add(int a, int b, int* c)
{
    *c = a + b;
}

// ChatGCP o1 pro: 20241223
/*
not using threadIdx.x or block indexing. This is because it’s a very simple example with only one block and one thread. No parallelization logic is needed for just a single thread.

*/
int main(int argc, char* argv[])
{
    int steps = (argc > 1) ? atoi(argv[1]) : 10000000000; // get command
    int terms = (argc > 2) ? atoi(argv[2]) : 10000;     // line arguments
    int threads = 512;
    //int blocks = (steps + threads - 1) / threads;  // ensure threads*blocks ≥ steps
    int c;        // Host variable to store the result
    int* d_c;     // Pointer to device memory

    time_t timeStart, timeEnd;
    double timeElapsed;

    time(&timeStart);

    // Allocate memory on the GPU
    hipMalloc((void**)&d_c, sizeof(int));
    
    // Launch the 'add' kernel on 1 block with 1 thread
    // kernelName<<<numBlocks, threadsPerBlock>>>(parameters...);
    add << <1, 1 >> > (2, 7, d_c);

    // Copy the result back from GPU to CPU
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    time(&timeEnd);
    timeElapsed = difftime(timeEnd, timeStart);

    // Print the result
    //std::cout << "2 + 7 = " << c << std::endl;
    printf("2 + 7 = %d duration: %.f\n",c, timeElapsed);

    // Free the GPU memory
    hipFree(d_c);

    return 0;
}


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);



/*
int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
*/
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
