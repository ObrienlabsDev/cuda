#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <time.h>

/**
* Michael O'Brien 20241223
* michael at obrienlabs.dev
* Collatz sequence running on NVidia GPUs like the RTX-3500 ada,A4000,A4500,4090 ada and A6000
* http://www.ericr.nl/wondrous/pathrecs.html
* https://github.com/obrienlabs/benchmark/blob/master/ObjectiveC/128bit/main.m
* https://github.com/obrienlabs/benchmark/blob/master/collatz_vs10/collatz_vs10/collatz_vs10.cpp
* https://github.com/ObrienlabsDev/cuda/blob/main/add_example/kernel_collatz.cu
* https://github.com/ObrienlabsDev/collatz/blob/main/src/main/java/dev/obrienlabs/collatz/service/CollatzUnitOfWork.java
* 
*/


/* CUDA Kernel runs on GPU device streaming core */
__global__ void addArrays(unsigned long long* a, unsigned long long* c, int N, unsigned long long iterations)
{
    // Calculate this thread's index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check boundary (in case N is not a multiple of blockDim.x)
    int path = 0;
    unsigned long long max = a[i];
    unsigned long long current = a[i];

    if (i < N)
    {
        // takes 130 sec on a mobile RTX-3500 ada 
        for (unsigned long q = 0; q < iterations; q++) {
            path = 0;
            max = a[i];
            current = a[i];

            do {
                path += 1;
                if (current % 2 == 0) {
                    current = current >> 1;
                }
                else {
                    current = 1 + current * 3;
                    if (current > max) {
                        max = current;
                    }
                }
            } while (current > 1);
        }
    }
    c[i] = max;
}

/* Host progrem */
int main(int argc, char* argv[])
{
    int cores = (argc > 1) ? atoi(argv[1]) : 5120; // get command
    const int N = 5120;
    int iterationPower = 22;
    unsigned long long iterations = 1 << iterationPower;
    // 256 threads per block is double the SM core count of 128 cores per SM:
    // 22, 256, 4096 = 130s
    // 22, 128, 4096 = 124
    // 22, 256, 5120 = 
    // 22, 128, 5120 = 125
    const int threadsPerBlock = 256;

    // Host arrays
    unsigned long long h_a[N];

    for (int q = 0; q < N; q++) {
        h_a[q] = 8528817511;
    }

    unsigned long long h_result[N] = { 0 };

    // Device pointers
    unsigned long long* d_a = nullptr;
    unsigned long long* d_c = nullptr;

    time_t timeStart, timeEnd;
    double timeElapsed;

    time(&timeStart);

    // Allocate memory on the GPU
    size_t size = N * sizeof(unsigned long long);
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_c, size);

    // Copy input data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);

    // Number of blocks = ceiling(N / threadsPerBlock)
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    printf("Iterations: %lld Threads: %d ThreadsPerBlock: %d Blocks: %d\n", iterations, N, threadsPerBlock, blocks);

    // Launch kernel
    // kernelName<<<numBlocks, threadsPerBlock>>>(parameters...);
    addArrays << <blocks, threadsPerBlock >> > (d_a, d_c, N, iterations);
    
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Copy result from device back to host
    hipMemcpy(h_result, d_c, size, hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "collatz:\n";
    for (int i = 0; i < N; i++)
    {
        std::cout << i << ": " << h_a[i] << " = " << h_result[i] << "\n";
    }

    time(&timeEnd);
    timeElapsed = difftime(timeEnd, timeStart);

    //std::cout << "2 + 7 = " << c << std::endl;
    printf("duration: %.f\n", timeElapsed);

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_c);

    return 0;
}

